#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;
using namespace std::chrono;

__global__ void matrixMultiply(double *A, double *B, double *C, int N) {
    __shared__ double tileA[32][32];
    __shared__ double tileB[32][32];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;

    for (int t = 0; t < (N + blockDim.x - 1)/blockDim.x; ++t) {
        if (row < N && (t*blockDim.x + threadIdx.x) < N) {
            tileA[threadIdx.y][threadIdx.x] = A[row*N + t*blockDim.x + threadIdx.x];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (col < N && (t*blockDim.y + threadIdx.y) < N) {
            tileB[threadIdx.y][threadIdx.x] = B[(t*blockDim.y + threadIdx.y)*N + col];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int k = 0; k < blockDim.x; ++k) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < N) {
        C[row*N + col] = sum;
    }
}

vector<vector<double>> read_matrix(const string &filename) {
    ifstream file(filename);
    vector<vector<double>> matrix;
    string line;
    while (getline(file, line)) {
        vector<double> row;
        size_t pos = 0;
        while (pos < line.size()) {
            size_t end_pos = line.find(' ', pos);
            if (end_pos == string::npos) end_pos = line.size();
            string num_str = line.substr(pos, end_pos - pos);
            double num = stod(num_str);
            row.push_back(num);
            pos = end_pos + 1;
        }
        matrix.push_back(row);
    }
    return matrix;
}

vector<vector<double>> multiply_matrices_cuda(const vector<vector<double>> &a, const vector<vector<double>> &b) {
    int N = a.size();
    double *h_A = new double[N*N];
    double *h_B = new double[N*N];
    double *h_C = new double[N*N];

    // Конвертация матриц в плоские массивы
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            h_A[i*N + j] = a[i][j];
            h_B[i*N + j] = b[i][j];
        }
    }

    
    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, N*N*sizeof(double));
    hipMalloc(&d_B, N*N*sizeof(double));
    hipMalloc(&d_C, N*N*sizeof(double));

    hipMemcpy(d_A, h_A, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N*N*sizeof(double), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);


    hipMemcpy(h_C, d_C, N*N*sizeof(double), hipMemcpyDeviceToHost);

    vector<vector<double>> result(N, vector<double>(N));
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            result[i][j] = h_C[i*N + j];
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return result;
}

void save_result(const vector<vector<double>> &result, const string &filename, double duration, double data_size) {
    ofstream file(filename);
    if (!file.is_open()) {
        cerr << "Could not open file " << filename << endl;
        exit(1);
    }
    for (const auto &row : result) {
        for (size_t j = 0; j < row.size(); ++j) {
            file << row[j];
            if (j != row.size() - 1) file << " ";
        }
        file << endl;
    }
    file << "Time taken: " << duration << " seconds" << endl;
    file << "Data size: " << data_size << " MB" << endl;
    file.close();
}

int main() {
    auto start = high_resolution_clock::now();

    auto matrix1 = read_matrix("matrix1.txt");
    auto matrix2 = read_matrix("matrix2.txt");

    auto multiply_start = high_resolution_clock::now();
    auto result = multiply_matrices_cuda(matrix1, matrix2);
    auto multiply_end = high_resolution_clock::now();

    double multiply_time = duration_cast<duration<double>>(multiply_end - multiply_start).count();

    size_t bytes = 2 * 1000 * 1000 * sizeof(double);
    double data_size = static_cast<double>(bytes) / (1024 * 1024);

    save_result(result, "result_cuda.txt", multiply_time, data_size);

    auto end = high_resolution_clock::now();
    double total_time = duration_cast<duration<double>>(end - start).count();
    cout << "Total execution time: " << total_time << " seconds" << endl;

    return 0;
}